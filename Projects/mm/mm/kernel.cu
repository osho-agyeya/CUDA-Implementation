#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
//matrix Addition
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include "kernel.h"

#define N 3
//int fin[N][N] = { 0 };
__global__ void MatAdd(int A[][N], int B[][N],int C[][N]) {
	int i = threadIdx.x;
	int j = threadIdx.y;

	C[i][j] = A[i][j] + B[i][j];
	/*
		#if __CUDA_ARCH__ >= 200
		atomicAdd(&C[i][j], A[i][j] + B[i][j]);
		#endif
		*/

}


int main() {

	int A[N][N] = { { 1,2,3 },{ 3,4,2 },{1,2,3 } };
	int B[N][N] = { { 5,6,1 },{ 7,8,2 },{1,2,3} };
	int C[N][N] = { { 0,0,0 },{ 0,0,0 },{0,0,0} };

	int(*pA)[N], (*pB)[N], (*pC)[N],(*pf)[N];

	hipMalloc((void**)&pA, (N*N) * sizeof(int));
	hipMalloc((void**)&pB, (N*N) * sizeof(int));
	hipMalloc((void**)&pC, (N*N) * sizeof(int));

	hipMemcpy(pA, A, (N*N) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pB, B, (N*N) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pC, C, (N*N) * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(pf, fin, (N*N) * sizeof(int), hipMemcpyHostToDevice);
	int numBlocks = 1;
	dim3 threadsPerBlock(N, N);
	MatAdd << <numBlocks, threadsPerBlock >>>(pA, pB, pC);

	hipMemcpy(C, pC, (N*N) * sizeof(int), hipMemcpyDeviceToHost);

	int i, j; printf("C = \n");
	for (i = 0; i<N; i++) {
		for (j = 0; j<N; j++) {
			printf("%d ", C[i][j]);
		}
		printf("\n");
	}

	hipFree(pA);
	hipFree(pB);
	hipFree(pC);

	printf("\n");

	return 0;
}