#include "hip/hip_runtime.h"
//matrix multiplication


#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#define N 3
__global__ void MatMul(int A[][N], int B[][N], int C[][N]) {
	int i = threadIdx.x;
	int j = threadIdx.y;
	int temp;
	for (int j1 = 0; j1 < N; j1++)
	{
		temp = A[i][j] * B[j][j1];
		#if __CUDA_ARCH__ >= 200
		atomicAdd(&C[i][j1], temp);
		#endif
	}
}


int main() {

	int A[N][N] = { { 1,2,3 },{ 4,5,6 },{ 7,8,9 } };
	int B[N][N] = { { 1,2,3 },{ 4,5,6 },{ 7,8,9 } };
	int C[N][N] = { { 0,0,0 },{ 0,0,0 },{ 0,0,0 } };

	int(*pA)[N], (*pB)[N], (*pC)[N], (*pf)[N];

	hipMalloc((void**)&pA, (N*N) * sizeof(int));
	hipMalloc((void**)&pB, (N*N) * sizeof(int));
	hipMalloc((void**)&pC, (N*N) * sizeof(int));

	hipMemcpy(pA, A, (N*N) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pB, B, (N*N) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pC, C, (N*N) * sizeof(int), hipMemcpyHostToDevice);
	int numBlocks = 1;
	dim3 threadsPerBlock(N,N);
	MatMul << <numBlocks, threadsPerBlock >> >(pA, pB, pC);

	hipMemcpy(C, pC, (N*N) * sizeof(int), hipMemcpyDeviceToHost);

	int i, j; printf("\nC = \n");
	for (i = 0; i<N; i++) {
		for (j = 0; j<N; j++) {
			printf("%d ", C[i][j]);
		}
		printf("\n");
	}

	hipFree(pA);
	hipFree(pB);
	hipFree(pC);

	printf("\n");

	return 0;
}