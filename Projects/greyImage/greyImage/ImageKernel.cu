#include "hip/hip_runtime.h"

//-----------------------------------------------------------------------------
// File: ImageKernel.cu
//
// Desc: CUDA kernel to convert RGBA image to gray.
//		 
//		 This kernel assumes 4-bytes per pixel RGBA image with channels Red, Green, Blue, and Alpha
//		 represented each by one byte (8-bits) and a range of values between 0 and 255 (2^8 - 1).
//
//		 Grey scale images are represented by a single intensity value per pixel 
//		 where each pixel is only 1 byte.
//
//		 Human eye perceives red, green, and blue colors unequally (humans are more sensitive to green and least to blue) 
//		 and for that reason we will use weighted formula (http://en.wikipedia.org/wiki/Grayscale):
//
//		I = 0.2126 * R + 0.7152 * G + 0.0722 * B
//		 
//		 
//
//-----------------------------------------------------------------------------

#include "stdafx.h"
#include "Image.h"
///////////////////////
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
//////////////////////

using namespace Bisque;

using std::ceilf;

#define N (1024 * 1024)				// blocks
#define BLOCK_WIDTH 32				// threads per block

// Converts RGBA image to gray scale intensity using the following formula:
// I = 0.2126 * R + 0.7152 * G + 0.0722 * B
__global__
void rgba_to_grayscale(unsigned char* const gray, const uchar4* const rgba, int rows, int cols)
{
	int r = blockIdx.y * blockDim.y + threadIdx.y;		// current row
	int c = blockIdx.x * blockDim.x + threadIdx.x;		// current column

	if ((r < rows) && (c < cols))
	{
		int idx = c + cols * r;		// current pixel index

		uchar4 pixel = rgba[idx];
		float intensity = 0.2126f * pixel.x + 0.7152f * pixel.y + 0.0722f * pixel.z;

		gray[idx] = (unsigned char)intensity;
	}
}

// Runs r8g8b8a8 to gray kernel
void RunRGBAtoGrayKernel(
	unsigned char*	gray,				// gray image: 1 byte per image --> this will be returned from this function
	uchar4*			rgba,				// rgba image: 4 bytes per image
	int				rows,				// image size: number of rows
	int				cols				// image size: number of columns
)
{
	const char* func = "RunGrayKernel";

	hipError_t hr = hipSuccess;

	int x = static_cast<int>(ceilf(static_cast<float>(cols) / BLOCK_WIDTH));
	int y = static_cast<int>(ceilf(static_cast<float>(rows) / BLOCK_WIDTH));

	const dim3 grid(x, y, 1);								// number of blocks
	const dim3 block(BLOCK_WIDTH, BLOCK_WIDTH, 1);			// block width: number of threads per block

	rgba_to_grayscale<<<grid, block>>>(gray, rgba, rows, cols);

	hr = hipDeviceSynchronize();																CHECK_CUDA_ERROR(hr, func, "rgba_to_grayscale failed.");
}
