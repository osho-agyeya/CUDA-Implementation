#include "hip/hip_runtime.h"
//ye nhi chla sort
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "Lock.h"

__global__ void addKernel(int *c)
{
	Lock ml;
	ml.lock();
    int i = threadIdx.x;
	int j = threadIdx.y;
	

	
	if ((i == j+1 && c[i] < c[j])|| (j == i+1 && c[i] > c[j]))
	{
			//exchange c[i] and c[j]
			//printf("c[i] : %d -- c[j] : %d\n", c[i], c[j]);


//#if __CUDA_ARCH__ >= 200
			int tempi = c[i];
			int tempj = c[j];
			c[i] = tempj;
			c[j] = tempi;
			//printf("i %d : j %d\n", i, j);
			//int z1 = atomicExch(&c[i], tempj);
			//int z2 = atomicExch(&c[j], tempi);
			//for (int i = 0; i < 5; i++)
			//{
			//	printf("%d ", c[i]);
			//}
			//printf("\n");
//#endif
			
	}
	ml.unlock();
}
hipError_t addWithCuda(int *a, unsigned int size)
{
	int *dev_a = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

	dim3 threadsPerBlock(size, size);
	addKernel << <1, threadsPerBlock >> >(dev_a);

	cudaStatus = hipGetLastError();
	
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(dev_a);


	return cudaStatus;
}

int main()
{
    const int arraySize = 5;
    int a[arraySize] = {5,4,3,2,1};

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(a,arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("this is the sorted array = {%d,%d,%d,%d,%d}\n",
        a[0], a[1], a[2], a[3], a[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.

