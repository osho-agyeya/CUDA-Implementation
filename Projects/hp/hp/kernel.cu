//histogram processing

#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t findFreqWithCuda(int *c, unsigned int fsize, const int *b, unsigned int size);




__global__ void addKernel(int *c, const int *b)
{
    int i = threadIdx.x;
	#if __CUDA_ARCH__ >= 200
	atomicAdd(&c[b[i]],1);
	#endif	
}

int main()
{
    const int arraySize = 11;
	const int fsize = 10005;//maximum value of the number in the array can be at max 10005
    const int a[arraySize] = { 1, 2, 3, 4, 5,1,1,1,2,3,6 };
	int maxx = INT_MIN;
	for (int i = 0; i < arraySize; i++)
	{
		//maxx = max(maxx, arr[i]);
		if (maxx < a[i])
			maxx=a[i];
	}
	int c[fsize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = findFreqWithCuda(c,fsize,a,arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "findFreqWithCuda failed!");
        return 1;
    }

	for (int i = 0; i <= maxx; i++)
	{
		if (c[i] != 0)
		{
			printf("%d occurs %d times\n", i, c[i]);
		}
	}

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t findFreqWithCuda(int *c, unsigned int fsize, const int *b, unsigned int size)
{
   // int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, fsize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
   

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	//dim3 threadsPerBlock(11,11,11);
    addKernel<<<1,size>>>(dev_c,dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, fsize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_b);
    
    return cudaStatus;
}
